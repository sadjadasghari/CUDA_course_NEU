#include <stdio.h>
#include <stdlib.h>
#include <hip/hip_runtime.h>
#include <time.h>

#define TIMER_CREATE(t)               \
  hipEvent_t t##_start, t##_end;     \
  hipEventCreate(&t##_start);        \
  hipEventCreate(&t##_end);               
 
 
#define TIMER_START(t)                \
  hipEventRecord(t##_start);         \
  hipEventSynchronize(t##_start);    \
 
 
#define TIMER_END(t)                             \
  hipEventRecord(t##_end);                      \
  hipEventSynchronize(t##_end);                 \
  hipEventElapsedTime(&t, t##_start, t##_end);  \
  hipEventDestroy(t##_start);                   \
  hipEventDestroy(t##_end);     
  
#define TILE_SIZE 16
#define CUDA_TIMING

unsigned char *input_gpu;
unsigned char *output_gpu;
unsigned int *histogram;
unsigned int *cumhistogram; 
unsigned int *SK;
double *PrRK;
double *alpha; 

double CLOCK() {
	struct timespec t;
	clock_gettime(CLOCK_MONOTONIC,  &t);
	return (t.tv_sec * 1000)+(t.tv_nsec*1e-6);
}

/*******************************************************/
/*                 Cuda Error Function                 */
/*******************************************************/
inline hipError_t checkCuda(hipError_t result) {
	#if defined(DEBUG) || defined(_DEBUG)
		if (result != cudaSuccess) {
			fprintf(stderr, "CUDA Runtime Error: %s\n", cudaGetErrorString(result));
			exit(-1);
		}
	#endif
		return result;
}
                
// Add GPU kernel and functions

///////////////////////////////////////////////////////////////////////////////////////////////////////////////
///////////////////////////////////////////////////////////////////////////////////////////////////////////////
///////////////////////////////////////////////////////////////////////////////////////////////////////////////
///////////////////////////////////////////////////////////////////////////////////////////////////////////////
__global__ void kernel(unsigned char *input,     //generates histogram
                       unsigned int *histogram){
    


  	int x = blockIdx.x*TILE_SIZE+threadIdx.x;
	int y = blockIdx.y*TILE_SIZE+threadIdx.y;
                
    	int location = y*TILE_SIZE*gridDim.x+x;
	int myItem = input[location];
	int myBin = myItem % 256;
	atomicAdd(&(histogram[myBin]),1);
    
	__syncthreads();
	//print histogram with its sum
  /*if(location==0)
		{
            int sum=0;
            for(int i=0;i<256;i++)
			{
				printf("%d %d \n",i,histogram[i]);
				sum+=histogram[i];
			}
            printf("sum=%d thredId=%d \n",sum,location);
		}
    */
}
///////////////////////////////////////////////////////////////////////////////////////////////////////////////
///////////////////////////////////////////////////////////////////////////////////////////////////////////////
///////////////////////////////////////////////////////////////////////////////////////////////////////////////
///////////////////////////////////////////////////////////////////////////////////////////////////////////////
__global__ void kernel2(unsigned int size, //generates probability
                       unsigned int *histogram,
                       double *PrRK){
    int x = blockIdx.x*TILE_SIZE+threadIdx.x;
    int y = blockIdx.y*TILE_SIZE+threadIdx.y;

    int ID = y*TILE_SIZE*gridDim.x+x;
    //int ID = blockIdx.x*TILE_SIZE+threadIdx.x;//check ID
    
    PrRK[ID]=(double)histogram[ID]/(double)size;
    //printf("PrRk is: %f\n", PrRK[ID]); 
}
///////////////////////////////////////////////////////////////////////////////////////////////////////////////
///////////////////////////////////////////////////////////////////////////////////////////////////////////////
///////////////////////////////////////////////////////////////////////////////////////////////////////////////
///////////////////////////////////////////////////////////////////////////////////////////////////////////////
__global__ void kernel3(double *alpha,
                        unsigned int *cumhistogram,
                        unsigned int *SK){
    int x = blockIdx.x*TILE_SIZE+threadIdx.x;
    int y = blockIdx.y*TILE_SIZE+threadIdx.y;
    printf("alpha:     %.15f \n", *alpha);
    int ID = y*TILE_SIZE*gridDim.x+x;
    //int ID = blockIdx.x*TILE_SIZE+threadIdx.x;//check ID
    SK[ID]=cumhistogram[ID]*(*alpha);
    printf("SK is : %d \n", SK[ID]); 
}
///////////////////////////////////////////////////////////////////////////////////////////////////////////////
///////////////////////////////////////////////////////////////////////////////////////////////////////////////
///////////////////////////////////////////////////////////////////////////////////////////////////////////////
///////////////////////////////////////////////////////////////////////////////////////////////////////////////
void histogram_gpu(unsigned char *data, 
                   unsigned int height, 
                   unsigned int width){
                         
	int gridXSize = 1 + (( width - 1) / TILE_SIZE);
	int gridYSize = 1 + ((height - 1) / TILE_SIZE);
	
	int XSize = gridXSize*TILE_SIZE;
	int YSize = gridYSize*TILE_SIZE;
	

	// Both are the same size (CPU/GPU).
	int size = XSize*YSize;
	
	// Allocate arrays in GPU memory
	checkCuda(hipMalloc((void**)&input_gpu     , size*sizeof(unsigned char)));
	checkCuda(hipMalloc((void**)&output_gpu    , size*sizeof(unsigned char)));
	checkCuda(hipMalloc((void**)&histogram     , 256*sizeof(unsigned int)));
    checkCuda(hipMalloc((void**)&cumhistogram  , 256*sizeof(unsigned int)));
    checkCuda(hipMalloc((void**)&PrRK          , 256*sizeof(double)));
    checkCuda(hipMalloc((void**)&SK            , 256*sizeof(unsigned int)));
    
    checkCuda(hipMemset(output_gpu ,   0, size*sizeof(unsigned char)));
    checkCuda(hipMemset(histogram,     0, 256*sizeof(unsigned int)));
    checkCuda(hipMemset(cumhistogram,     0, 256*sizeof(unsigned int)));
    checkCuda(hipMemset(PrRK,          0, 256*sizeof(double)));
    checkCuda(hipMemset(SK,            0, 256*sizeof(unsigned int)));
    
        // Copy data to GPU
    checkCuda(hipMemcpy(input_gpu,
                         data,
                         size*sizeof(char),
                         hipMemcpyHostToDevice));

	checkCuda(hipDeviceSynchronize());
    
    // Execute algorithm
        
	dim3 dimGrid(gridXSize, gridYSize);
    dim3 dimBlock(TILE_SIZE, TILE_SIZE);
    
    dim3 dimGrid2(1, 1);
    dim3 dimBlock2(TILE_SIZE, TILE_SIZE);
    
    dim3 dimGrid3(1, 1);
    dim3 dimBlock3(TILE_SIZE, TILE_SIZE);
    
    // Kernel Call
	#if defined(CUDA_TIMING)
		float Ktime;
		TIMER_CREATE(Ktime);
		TIMER_START(Ktime);
	#endif
       	//printf("histogrammm: %d\n %d\n", histogram[250], histogram[0]); 
        kernel<<<dimGrid, dimBlock>>>(input_gpu, 
                                      histogram);
        double *alpha2 = new(double);
	*alpha2= 255/(double)size;
 
	checkCuda(hipDeviceSynchronize());
    
    kernel2<<<dimGrid2, dimBlock2>>>(size,
                                     histogram,
                                     PrRK);
    
    
    
    // generate cumhistogram
    unsigned int *cum_histogram;
    cum_histogram= new(unsigned int[256]);
    unsigned int *histogram2;
        histogram2 = new(unsigned int[256]);
        checkCuda(hipMemcpy(histogram2,
                        histogram,
                        256*sizeof(unsigned int),
                        hipMemcpyDeviceToHost));
    //printf("histogram2 is: %d\n", histogram2[100]);
    /*unsigned int *histogram2;
        histogram2 = new(unsigned int[256]);
        checkCuda(cudaMemcpy(histogram2,
                        histogram,
                        256*sizeof(unsigned int),
                        cudaMemcpyDeviceToHost));*/
    cum_histogram[0]=histogram2[0];
    for (int i=1;i<256;i++)
    {
        cum_histogram[i]=cum_histogram[i-1]+histogram2[i];
    	  //printf("cum_histogram: %d \n", cum_histogram[i]);
    }
    checkCuda(hipDeviceSynchronize());
    
    checkCuda(hipMemcpy(cumhistogram,
                        cum_histogram,
                        256*sizeof(unsigned int),
                        hipMemcpyHostToDevice));
    checkCuda(hipMemcpy(alpha,
	                alpha2,
                        sizeof(double),
                        hipMemcpyHostToDevice));
    //checkCuda(cudaMemset(cumhistogram,cum_histogram, 256*sizeof(unsigned int)));
    kernel3<<<dimGrid2, dimBlock2>>>(alpha,
                                     cumhistogram,
                                     SK);
    
    
	//printf("gridXsize is: %d\n", gridXSize);
	//printf("gridYsize is: %d\n", gridYSize);
        //printf("TILE_SIZE is: %d\n",TILE_SIZE);
	printf("alpha:     %.15f \n", *alpha2);                                   
	//printf("%d \n", histogram[0]);
	double *PrRK2;
	PrRK2 = new(double[256]);
	checkCuda(hipMemcpy(PrRK2,
                        PrRK,
                        256*sizeof(double),
                        hipMemcpyDeviceToHost));
        double sum =0; 
	for (int i=0; i<256; i++)
  {
        	//printf("%f \n", PrRK2[i]);
       		sum+=PrRK2[i];
	}
	printf ("real PrRK2 sum is: %f\n", sum);
        checkCuda(hipDeviceSynchronize());
	
	#if defined(CUDA_TIMING)
		TIMER_END(Ktime);
		printf("Kernel Execution Time: %f ms\n", Ktime);
	#endif
        
	// Retrieve results from the GPU
	checkCuda(hipMemcpy(data, 
			output_gpu, 
			size*sizeof(unsigned char), 
			hipMemcpyDeviceToHost));
                        
        // Free resources and end the program
	checkCuda(hipFree(output_gpu));
	checkCuda(hipFree(input_gpu));
	checkCuda(hipFree(histogram));
}
////////////////////////////////////////////////////////////
////////////////////////////////////////////////////////////
////////////////////////////////////////////////////////////
////////////////////////////////////////////////////////////
void histogram_gpu_warmup(unsigned char *data,
                   unsigned int height, 
                   unsigned int width){
                         
	int gridXSize = 1 + (( width - 1) / TILE_SIZE);
	int gridYSize = 1 + ((height - 1) / TILE_SIZE);
	
	int XSize = gridXSize*TILE_SIZE;
	int YSize = gridYSize*TILE_SIZE;
	
	// Both are the same size (CPU/GPU).
	int size = XSize*YSize;
	
	// Allocate arrays in GPU memory
	checkCuda(hipMalloc((void**)&input_gpu   , size*sizeof(unsigned char)));
	checkCuda(hipMalloc((void**)&output_gpu  , size*sizeof(unsigned char)));
    checkCuda(hipMalloc((void**)&histogram , 256*sizeof(unsigned int)));
   
    
        checkCuda(hipMemset(output_gpu , 0 , size*sizeof(unsigned char)));
		checkCuda(hipMemset(histogram, 0, 256*sizeof(unsigned int)));

				
        // Copy data to GPU
        checkCuda(hipMemcpy(input_gpu, 
			data, 
			size*sizeof(char), 
			hipMemcpyHostToDevice));

	checkCuda(hipDeviceSynchronize());
        
        // Execute algorithm
        
	dim3 dimGrid(gridXSize, gridYSize);
        dim3 dimBlock(TILE_SIZE, TILE_SIZE);
        
        kernel<<<dimGrid, dimBlock>>>(input_gpu, 
                                      histogram);
                                             
        checkCuda(hipDeviceSynchronize());
        
	// Retrieve results from the GPU
	checkCuda(hipMemcpy(data, 
			output_gpu, 
			size*sizeof(unsigned char), 
			hipMemcpyDeviceToHost));
                        
        // Free resources and end the program
	checkCuda(hipFree(output_gpu));
	checkCuda(hipFree(input_gpu));

}

