
#include <stdio.h>
#include <stdlib.h>
#include <hip/hip_runtime.h>
#include <time.h>

#define TIMER_CREATE(t)               \
  hipEvent_t t##_start, t##_end;     \
  hipEventCreate(&t##_start);        \
  hipEventCreate(&t##_end);               
 
 
#define TIMER_START(t)                \
  hipEventRecord(t##_start);         \
  hipEventSynchronize(t##_start);    \
 
 
#define TIMER_END(t)                             \
  hipEventRecord(t##_end);                      \
  hipEventSynchronize(t##_end);                 \
  hipEventElapsedTime(&t, t##_start, t##_end);  \
  hipEventDestroy(t##_start);                   \
  hipEventDestroy(t##_end);     
  
#define TILE_SIZE 16
#define CUDA_TIMING

unsigned char *input_gpu;
unsigned char *output_gpu;
unsigned int *histogram;

double CLOCK() {
	struct timespec t;
	clock_gettime(CLOCK_MONOTONIC,  &t);
	return (t.tv_sec * 1000)+(t.tv_nsec*1e-6);
}

/*******************************************************/
/*                 Cuda Error Function                 */
/*******************************************************/
inline hipError_t checkCuda(hipError_t result) {
	#if defined(DEBUG) || defined(_DEBUG)
		if (result != cudaSuccess) {
			fprintf(stderr, "CUDA Runtime Error: %s\n", cudaGetErrorString(result));
			exit(-1);
		}
	#endif
		return result;
}
                
// Add GPU kernel and functions
__global__ void kernel(unsigned char *input, 
                       unsigned int *histogram){
    


  	int x = blockIdx.x*TILE_SIZE+threadIdx.x;
	int y = blockIdx.y*TILE_SIZE+threadIdx.y;
                
        int location = 	y*TILE_SIZE*gridDim.x+x;
	int myItem = input[location];
	int myBin = myItem % 256;
	atomicAdd(&(histogram[myBin]),1);
       // output[location] = x%255;
	__syncthreads();
	//printf("!!!!!!!!!!!!!!\n");	
	if(location==0)
		{int sum=0;
		 for(int i=0;i<256;i++)
			{
				printf("%d %d \n",i,histogram[i]);
				sum+=histogram[i];
			}
		printf("sum=%d thredId=%d \n",sum,location);
		}
	//printf("%d ", myBin);
}

void histogram_gpu(unsigned char *data, 
                   unsigned int height, 
                   unsigned int width){
                         
	int gridXSize = 1 + (( width - 1) / TILE_SIZE);
	int gridYSize = 1 + ((height - 1) / TILE_SIZE);
	
	int XSize = gridXSize*TILE_SIZE;
	int YSize = gridYSize*TILE_SIZE;
	

	// Both are the same size (CPU/GPU).
	int size = XSize*YSize;
	
	// Allocate arrays in GPU memory
	checkCuda(hipMalloc((void**)&input_gpu   , size*sizeof(unsigned char)));
	checkCuda(hipMalloc((void**)&output_gpu  , size*sizeof(unsigned char)));
	checkCuda(hipMalloc((void**)&histogram , 256*sizeof(unsigned int)));

        checkCuda(hipMemset(output_gpu , 0 , size*sizeof(unsigned char)));
		checkCuda(hipMemset(histogram, 0, 256*sizeof(unsigned int)));
        // Copy data to GPU
        checkCuda(hipMemcpy(input_gpu, 
			data, 
			size*sizeof(char), 
			hipMemcpyHostToDevice));

	checkCuda(hipDeviceSynchronize());
        
        // Execute algorithm
        
	dim3 dimGrid(gridXSize, gridYSize);
        dim3 dimBlock(TILE_SIZE, TILE_SIZE);

        // Kernel Call
	#if defined(CUDA_TIMING)
		float Ktime;
		TIMER_CREATE(Ktime);
		TIMER_START(Ktime);
	#endif
       	//printf("histogrammm: %d\n %d\n", histogram[250], histogram[0]); 
        kernel<<<dimGrid, dimBlock>>>(input_gpu, 
                                      histogram);
        double alpha = 255/size;
	checkCuda(hipDeviceSynchronize());
	//printf("gridXsize is: %d\n", gridXSize);
	//printf("gridYsize is: %d\n", gridYSize);
        //printf("TILE_SIZE is: %d\n",TILE_SIZE);
	printf("alpha:     %.15f \n", alpha);                                   
	//printf("%d \n", histogram[0]);
	unsigned int *histogram2;
	histogram2 = new(unsigned int[256]);
	checkCuda(hipMemcpy(histogram2,
                        histogram,
                        256*sizeof(unsigned int),
                        hipMemcpyDeviceToHost));
        int sum =0; 
	for (int i=0; i<256; i++)
        {
        	printf("%d \n", histogram2[i]);
       		sum+=histogram2[i];
	}
	printf ("real sum is: %d\n", sum);
        checkCuda(hipDeviceSynchronize());
	
	#if defined(CUDA_TIMING)
		TIMER_END(Ktime);
		printf("Kernel Execution Time: %f ms\n", Ktime);
	#endif
        
	// Retrieve results from the GPU
	checkCuda(hipMemcpy(data, 
			output_gpu, 
			size*sizeof(unsigned char), 
			hipMemcpyDeviceToHost));
                        
        // Free resources and end the program
	checkCuda(hipFree(output_gpu));
	checkCuda(hipFree(input_gpu));
	checkCuda(hipFree(histogram));
}

void histogram_gpu_warmup(unsigned char *data, 
                   unsigned int height, 
                   unsigned int width){
                         
	int gridXSize = 1 + (( width - 1) / TILE_SIZE);
	int gridYSize = 1 + ((height - 1) / TILE_SIZE);
	
	int XSize = gridXSize*TILE_SIZE;
	int YSize = gridYSize*TILE_SIZE;
	
	// Both are the same size (CPU/GPU).
	int size = XSize*YSize;
	
	// Allocate arrays in GPU memory
	checkCuda(hipMalloc((void**)&input_gpu   , size*sizeof(unsigned char)));
	checkCuda(hipMalloc((void**)&output_gpu  , size*sizeof(unsigned char)));
		checkCuda(hipMalloc((void**)&histogram , 256*sizeof(unsigned int)));

        checkCuda(hipMemset(output_gpu , 0 , size*sizeof(unsigned char)));
		checkCuda(hipMemset(histogram, 0, 256*sizeof(unsigned int)));

				
        // Copy data to GPU
        checkCuda(hipMemcpy(input_gpu, 
			data, 
			size*sizeof(char), 
			hipMemcpyHostToDevice));

	checkCuda(hipDeviceSynchronize());
        
        // Execute algorithm
        
	dim3 dimGrid(gridXSize, gridYSize);
        dim3 dimBlock(TILE_SIZE, TILE_SIZE);
        
        kernel<<<dimGrid, dimBlock>>>(input_gpu, 
                                      histogram);
                                             
        checkCuda(hipDeviceSynchronize());
        
	// Retrieve results from the GPU
	checkCuda(hipMemcpy(data, 
			output_gpu, 
			size*sizeof(unsigned char), 
			hipMemcpyDeviceToHost));
                        
        // Free resources and end the program
	checkCuda(hipFree(output_gpu));
	checkCuda(hipFree(input_gpu));

}

